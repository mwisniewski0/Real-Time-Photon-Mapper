#include "hip/hip_runtime.h"
﻿#include "Renderer.h"
#include "Camera.h"
#include "../cuda_common/helpers.h"
#include "../common/geometry.h"
#include <iostream>
#include "../cuda_common/gpuBvh.h"
#include "../common/ply.h"
#include "cudaRenderer.h"
#include "GlHelp.h"
#include "sdlHelpers.h"
#include "../obj_parser/obj_file_parser.h"
#include <fstream>


// Photon-defined SDL messages
const auto UPDATE_TIMER = 100;
const auto FPS_COUNTER_TIMER = 101;


void UserMovementInfo::updateCamera(Camera* camera)
{
	if (movingForward)
	{
		camera->moveForward(0.02);
	}
	if (movingBackward)
	{
		camera->moveBackward(0.02);
	}
	if (movingLeft)
	{
		camera->moveLeft(0.02);
	}
	if (movingRight)
	{
		camera->moveRight(0.02);
	}
}

void FpsCounter::secondPassed()
{
	int FPS = framesRendered - lastFrame;
	std::cout << FPS << "\n";
	lastFrame = framesRendered;
}

void FpsCounter::frameRendered()
{
	framesRendered += 1;
}

void Renderer::initGL()
{
	// Initialize SDL's Video subsystem
	if (SDL_Init(SDL_INIT_VIDEO | SDL_INIT_TIMER) < 0)
	{
		throw std::runtime_error("SDL could not be initialized");
	}

	mainWindow = SDL_CreateWindow(config.windowTitle.c_str(), SDL_WINDOWPOS_CENTERED,
		SDL_WINDOWPOS_CENTERED, config.outputWidth, config.outputHeight, SDL_WINDOW_OPENGL);
	if (!mainWindow)
	{
		throw std::runtime_error("Unable to create window");
	}

	// Create our opengl context and attach it to our window
	mainContext = SDL_GL_CreateContext(mainWindow);

	SDL_GL_SetAttribute(SDL_GL_CONTEXT_PROFILE_MASK, SDL_GL_CONTEXT_PROFILE_CORE);
	SDL_GL_SetAttribute(SDL_GL_CONTEXT_MAJOR_VERSION, 4);
	SDL_GL_SetAttribute(SDL_GL_CONTEXT_MINOR_VERSION, 6);

	// TODO: consider whether we want double buffering or not. We only draw one rectangle, so we
	// might not need the extra buffer.
	SDL_GL_SetAttribute(SDL_GL_DOUBLEBUFFER, 1);
	SDL_GL_SetSwapInterval(0);

	glewExperimental = GL_TRUE;
	glewInit();

	glProgramId = glhelp::createShaderProgram(
		readFileToString(config.vtxShaderPath), readFileToString(config.fragShaderPath));
	glUseProgram(glProgramId);
}

struct Triangle2
{
	float3 a, b, c;
	unsigned materialIndex;

	Triangle toTriangle()
	{
		return Triangle::from3Points(a, b, c, materialIndex);
	}
};

Scene Renderer::loadModel()
{
	// auto file = std::ifstream("C::\\test\\shitsies.photon", std::ios_base::binary | std::ios_base::in);
	// return readFromStream<Scene>(file);
	auto scene = loadObj("C:\\Users\\m_wis\\Documents\\3dsMax\\export\\vonia\\vonia.obj",
		"C:\\Users\\m_wis\\Documents\\3dsMax\\export\\vonia\\",
		"");
	// TODO:
	scene.lights.push_back(PointLightSource{
		{ 1.22f, 4.10f, -0.70f },
		{ 0.52f, 0.52f, 0.62f }
	});
	scene.lights.push_back(PointLightSource{
		{ 3.16f, 4.55f, -1.28f },
		{ 0.62f, 0.52f, 0.52f }
	});

	scene.spheres.push_back(Sphere{
		{0.207f, 1.656f, -3.006f},
		0.2f,
		{
			{ 0.0f,0.0f,0.0f },
			{ 0.0f,0.0f,0.0f },
			{ 0.9f, 0.9f, 0.9f },
			1.0f,
			1.6f,
			false,
			{}
		}
	});

	return scene;
	// auto file = std::ofstream("C:\\test\\shitsies.photon", std::ios_base::binary | std::ios_base::out);
	// writeToStream(file, scene);
	// return scene;
	// return loadObj("C:/Code/photon/renderer/sdlTest/sdlTest/stage2_cuda/models/bunny.obj",
	// 	"C:/Code/photon/renderer/sdlTest/sdlTest/stage2_cuda/models/",
	// 	"C:/Code/photon/renderer/sdlTest/sdlTest/stage2_cuda/models/");
// 	// TODO(#2): this function will load the model stored in config.inputFile. Currently, the model
// 	// is hardcoded (our model loading code is still in production).
// 	
// 	std::vector<Triangle> triangles; // = loadTriangles("models/dragon_vrip.ply", Material{ {0,1,1}, {0.8f, 0.8f, 0.8f}, 1.4f, 0x0000 });
//
// 	std::vector<MaterialInfo> materials(5);
// 	materials[0] = { {
// 		{ 1.f,1.f,1.f },
// 		{ 0.f,0.f,0.f },
// 		{ 0.f,0.f,0.f },
// 		0.f,
// 		1.f,
//
// 		true,
// 		{}
// 		}, "models/bricks.png"
// 	};
// 	materials[1] = { {
// 		{ 1.f,1.f,1.f },
// 		{ 0.f,0.f,0.f },
// 		{ 0.f,0.f,0.f },
// 		0.f,
// 		1.f,
//
// 		true,
// 		{}
// 		}, "models/asphalt.png"
// 	};
// 	materials[2] = { {
// 		{ 1.f,1.f,1.f },
// 		{ 0.f,0.f,0.f },
// 		{ 0.f,0.f,0.f },
// 		0.f,
// 		1.f,
//
// 		false,
// 		{}
// 		}
// 	};
// 	materials[3] = { {
// 		{ 0.f,0.f,0.f },
// 		{ 0.9f,0.9f,0.9f },
// 		{ 0.f,0.f,0.f },
// 		0.f,
// 		1.f,
//
// 		false,
// 		{}
// 		}
// 	};
// 	materials[4] = { {
// 		{ 0.f,0.f,0.f },
// 		{ 0.0f,0.0f,0.0f },
// 		{ 0.9f,0.9f,0.9f },
// 		0.f,
// 		1.6f,
//
// 		false,
// 		{}
// 		}
// 	};
// 	auto bricks = GPUTexture::fromPng("models/bricks.png");
// 	auto asphalt = GPUTexture::fromPng("models/asphalt.png");
//
// 	Scene scene;
//
// 	Triangle2 t;
//
// 	// Back wall
// 	t.a = { -1, 1, 1 };
// 	t.b = { 1, 1, 1 };
// 	t.c = { -1, -1, 1 };
// 	t.materialIndex = 3;
// 	auto tri = t.toTriangle();
// 	tri.v0vt = make_float3(0.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
//
// 	t.a = { 1, 1, 1 };
// 	t.b = { 1, -1, 1 };
// 	t.c = { -1, -1, 1 };
// 	t.materialIndex = 3;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
// 	// Front wall
// 	t.a = { -1, 1, -4 };
// 	t.b = { 1, 1, -4 };
// 	t.c = { -1, -1, -4 };
// 	t.materialIndex = 0;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(0.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
// 	t.a = { 1, 1, -4 };
// 	t.b = { 1, -1, -4 };
// 	t.c = { -1, -1, -4 };
// 	t.materialIndex = 0;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
// 	// Left wall
// 	t.a = { -1, -1, 1 };
// 	t.b = { -1, 1, 1 };
// 	t.c = { -1, -1, -4 };
// 	t.materialIndex = 0;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
// 	t.a = { -1, 1, 1 };
// 	t.b = { -1, 1, -4 };
// 	t.c = { -1, -1, -4 };
// 	t.materialIndex = 0;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
//
// 	// Right wall
// 	t.a = { 1, -1, 1 };
// 	t.b = { 1, 1, 1 };
// 	t.c = { 1, -1, -4 };
// 	t.materialIndex = 0;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
// 	t.a = { 1, 1, 1 };
// 	t.b = { 1, 1, -4 };
// 	t.c = { 1, -1, -4 };
// 	t.materialIndex = 0;
// 	triangles.push_back(t.toTriangle());
//
//
// 	// Top wall
// 	t.a = { -1, 1, 1 };
// 	t.b = { 1, 1, 1 };
// 	t.c = { -1, 1, -4 };
// 	t.materialIndex = 2;
// 	triangles.push_back(t.toTriangle());
//
//
// 	t.a = { 1, 1, 1 };
// 	t.b = { 1, 1, -4 };
// 	t.c = { -1, 1, -4 };
// 	t.materialIndex = 2;
// 	triangles.push_back(t.toTriangle());
//
//
// 	// Bottom wall
// 	t.a = { -1, -1, 1 };
// 	t.b = { 1, -1, 1 };
// 	t.c = { -1, -1, -4 };
// 	t.materialIndex = 1;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
//
// 	t.a = { 1, -1, 1 };
// 	t.b = { 1, -1, -4 };
// 	t.c = { -1, -1, -4 };
// 	t.materialIndex = 1;
// 	tri = t.toTriangle();
// 	tri.v0vt = make_float3(1.0f, 0.0f, 0.0f);
// 	tri.v1vt = make_float3(1.0f, 1.0f, 0.0f);
// 	tri.v2vt = make_float3(0.0f, 1.0f, 0.0f);
// 	triangles.push_back(tri);
//
// //
// //	// prism
// //	t.a = { 0, 0.5, 0 }; // A
// //	t.b = { 0, 0, 0.5 }; // B
// //	t.c = { -0.5, 0, 0 }; // C
// //	t.material = {
// //		{ 1, 1, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// // TODO: add  triangles.push_back(t);
// //	t.a = { 0, 0.5, 0 }; // A
// //	t.b = { -0.5, 0, 0 }; // C
// //	t.c = { 0, 0, -0.5 }; // D
// //	t.material = {
// //		{ 1, 0, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// //	t.a = { 0, 0.5, 0 }; // A
// //	t.b = { 0, 0, -0.5 }; // D
// //	t.c = { 0.5, 0, 0 }; // E
// //	t.material = {
// //		{ 0, 0, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// //	t.a = { 0, 0.5, 0 }; // A
// //	t.b = { 0.5, 0, 0 }; // E
// //	t.c = { 0, 0, 0.5 }; // B
// //	t.material = {
// //		{ 1, 0, 0 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// //
// //
// //	t.a = { 0, -0.5, 0 }; // F
// //	t.b = { 0, 0, 0.5 }; // B
// //	t.c = { -0.5, 0, 0 }; // C
// //	t.material = {
// //		{ 1, 0, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// //	t.a = { 0, -0.5, 0 }; // F
// //	t.b = { -0.5, 0, 0 }; // C
// //	t.c = { 0, 0, -0.5 }; // D
// //	t.material = {
// //		{ 1, 0, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// //	t.a = { 0, -0.5, 0 }; // F
// //	t.b = { 0, 0, -0.5 }; // D
// //	t.c = { 0.5, 0, 0 }; // E
// //	t.material = {
// //		{ 1, 0, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
// //	t.a = { 0, -0.5, 0 }; // F
// //	t.b = { 0.5, 0, 0 }; // E
// //	t.c = { 0, 0, 0.5 }; // B
// //	t.material = {
// //		{ 1, 0, 1 }, // color
// //		{ 0.8f, 0.8f, 0.8f }, // reflectivity
// //		2.5,  // refractive index (diamond)
// //		0x0002 // type
// //	};
//
// 	scene.spheres.emplace_back();
// 	scene.spheres[scene.spheres.size() - 1] = {
// 		{ -0.5f, 0.3f, 0 }, 0.2f,
// 		{
// 			{ 0.0f, 0.0f, 0.0f }, // diffuse
// 			{ 0.89f, 0.89f, 0.89f }, // specular
// 			{ 0.0f, 0.0f, 0.f }, // transmission
// 			1.0f,  // shininess
// 			2.5f,  // refractive index (diamond)
// 			false
// 		}
// 	};
// 	// scene.spheres.emplace_back();
// 	// scene.spheres[scene.spheres.size() - 1] = {
// 	// 	{ 0, -0.3f, 0 }, 0.2f,
// 	// 	{
// 	// 		{ 0.0f, 0.0f, 0.0f }, // diffuse
// 	// 		{ 0.0f, 0.0f, 0.0f }, // specular
// 	// 		{ 0.9f, 0.9f, 0.9f }, // transmission
// 	// 		1.0f,  // shininess
// 	// 		1.7f,  // refractive index (diamond)
// 	// 		false
// 	// 	}
// 	// };
// 	scene.spheres.emplace_back();
// 	scene.spheres[scene.spheres.size() - 1] = {
// 		{ 0.5f, 0.3f, 0 }, 0.2f,
// 		{
// 			{ 0.0f, 0.0f, 0.0f }, // diffuse
// 			{ 0.89f, 0.89f, 0.89f }, // specular
// 			{ 0.f, 0.f, 0.f }, // transmission
// 			1.0f,  // shininess
// 			2.5f,  // refractive index (diamond)
// 			false
// 		},
// 	};
// 	scene.triangleData = std::move(*buildBVH(std::move(triangles))->toRaw());
//
// 	scene.lights.push_back(PointLightSource{
// 		{0, 0.8f, 0},
// 		{1, 1, 1},
// 		});
//
// 	scene.materials = std::move(materials);
//
// 	return scene;
}

float degToRadians(float a)
{
	return 0.017453292 * a;
}

Renderer::Renderer(const RendererConfig& config)
{
	this->config = config;

	// TODO(#2): The input file should specify information about camera. This will be hardcoded for
	// now.
	// this->camera = Camera::fromHorizontalFov(make_float3(0, 0, -10), make_float3(0, 0, 1),
	// 	make_float3(0, 1, 0), degToRadians(config.horizontalFovDegrees),
	// 	((float)config.outputWidth) / config.outputHeight);


	this->camera = Camera::fromHorizontalFov(make_float3(1.369, 3.40, -0.40), make_float3(1.37, -3.52, -8.94),
		make_float3(8.609e-05, 0.7772, -0.6296), degToRadians(config.horizontalFovDegrees),
		((float)config.outputWidth) / config.outputHeight);
}

void Renderer::renderFrame()
{
	glClearColor(0.5, 0.5, 0.5, 1);
	glClear(GL_COLOR_BUFFER_BIT);
	glhelp::drawSquare(0, 0, 2);
	SDL_GL_SwapWindow(mainWindow);
}

void Renderer::loop()
{
	CudaRenderer r(config.outputWidth, config.outputHeight);
	r.loadScene(loadModel());

	bool loop = true;

	// Update 20 times per second
	registerPeriodicalSDLMessage(1000. / 35., UPDATE_TIMER);
	registerPeriodicalSDLMessage(1000., FPS_COUNTER_TIMER);


	while (loop)
	{
		SDL_Event event;
		while (SDL_PollEvent(&event))
		{
			if (event.type == SDL_QUIT)
				loop = false;

			if (event.type == SDL_KEYDOWN)
			{
				switch (event.key.keysym.sym)
				{
				case SDLK_ESCAPE:
					loop = false;
					break;
				case SDLK_w:
					userInputState.movingForward = true;
					break;
				case SDLK_s:
					userInputState.movingBackward = true;
					break;
				case SDLK_a:
					userInputState.movingLeft = true;
					break;
				case SDLK_d:
					userInputState.movingRight = true;
					break;
				}
			}

			if (event.type == SDL_KEYUP)
			{
				switch (event.key.keysym.sym)
				{
				case SDLK_w:
					userInputState.movingForward = false;
					break;
				case SDLK_s:
					userInputState.movingBackward = false;
					break;
				case SDLK_a:
					userInputState.movingLeft = false;
					break;
				case SDLK_d:
					userInputState.movingRight = false;
					break;
				}
			}

			if (event.type == SDL_MOUSEMOTION)
			{
				SDL_SetRelativeMouseMode(SDL_TRUE);
				camera->absTurnRight(event.motion.xrel * 0.001);
				camera->absTurnUp(event.motion.yrel * 0.001);
			}

			if (event.type == SDL_USEREVENT)
			{
				switch (event.user.code)
				{
				case UPDATE_TIMER:
					userInputState.updateCamera(camera.get());
					break;
				case FPS_COUNTER_TIMER:
					fpsCounter.secondPassed();
					break;
				}
			}
		}

		r.renderFrame(*camera);
		SDL_GL_SwapWindow(mainWindow);
		fpsCounter.frameRendered();
	}
}

void Renderer::cleanup()
{
	SDL_GL_DeleteContext(mainContext);
	SDL_DestroyWindow(mainWindow);
	SDL_Quit();
}

void Renderer::run()
{
	initGL();
	loop();
	cleanup();
}
