#include "hip/hip_runtime.h"
#include "gpuBvh.h"


struct BVHGpuDataRaw
{
	std::vector<GpuBvhNode> bvhNodes;
	std::vector<Triangle> triangles;

	BVHGpuData toGpu() const
	{
		BVHGpuData data;
		data.triangles = vectorToGpu(triangles);
		data.bvhNodes = vectorToGpu(bvhNodes);
		return data;
	}
};


void makeGpuBvhInternal(const BVHNode* node, BVHGpuDataRaw* data)
{
	data->bvhNodes.emplace_back();
	int currIndex = data->bvhNodes.size() - 1;
	auto box = node->boundingBox;
	data->bvhNodes[currIndex].setBoundingBox(box);

	if (node->isLeaf())
	{
		const BVHLeaf* leaf = dynamic_cast<const BVHLeaf*>(node);
		data->bvhNodes[currIndex].setAsLeaf();
		data->bvhNodes[currIndex].setCount(leaf->triangles.size());
		data->bvhNodes[currIndex].u.leaf.offset = data->triangles.size();

		for (const auto& shape : leaf->triangles)
		{
			data->triangles.push_back(shape);
		}
	}
	else
	{
		const BVHInner* inner = dynamic_cast<const BVHInner*>(node);
		data->bvhNodes[currIndex].u.inner.left = data->bvhNodes.size(); // The next node will be the left node
		makeGpuBvhInternal(inner->left.get(), data);

		data->bvhNodes[currIndex].u.inner.right = data->bvhNodes.size(); // The next node will be the right node
		makeGpuBvhInternal(inner->right.get(), data);
	}
}

void GpuBvhNode::setBoundingBox(BoundingBox box)
{
	min = box.minCoords;
	max = box.maxCoords;
}


BVHGpuData makeGpuBvh(const BVHNode* root)
{
	BVHGpuDataRaw data;
	makeGpuBvhInternal(root, &data);
	return data.toGpu();
}
